#include "hip/hip_runtime.h"

/* lenet.cu */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "header.h"

#define IMAGE_FILE       "./txt/image1000/"
#define CHECK_PARAMS    (0)

#define IMAGE_SIZE      (1 * 28 * 28)

#define CONV1_W_SIZE    (20 * 1 * 5 * 5)
#define CONV1_B_SIZE    (20)
#define CONV1_OUT_SIZE  (20 * 24 * 24)

#define POOL1_OUT_SIZE  (20 * 12 * 12)

#define CONV2_W_SIZE    (50 * 20 * 5 * 5)
#define CONV2_B_SIZE    (50)
#define CONV2_OUT_SIZE  (50 * 8 * 8)

#define POOL2_OUT_SIZE  (50 * 4 * 4)

#define FC1_W_SIZE      (500 * 800)
#define FC1_B_SIZE      (500)
#define FC1_OUT_SIZE    (500)

#define FC2_W_SIZE      (10 * 500)
#define FC2_B_SIZE      (10)
#define FC2_OUT_SIZE    (10)

#define CUDA_SAFE_CALL(call)                                                \
    do {                                                                    \
        hipError_t err = (call);                                           \
                                                                            \
        if (err != hipSuccess) {                                           \
            fprintf(stderr, "Error (%s:%d), code: %d, reason: %s\n",        \
                    __FILE__, __LINE__, err, hipGetErrorString(err));      \
            exit(EXIT_FAILURE);                                             \
        }                                                                   \
    } while (0)

void check_result(float* hostResult, float* gpuResult, int size)
{
    int i;

    for (i = 0; i < size; ++i) {
        if (fabs(hostResult[i] - gpuResult[i]) > 5.0e-2) {
            printf("check_result() failed at index %d\n", i);
            printf("GPU result: %f, Host result: %f\n",
                   gpuResult[i], hostResult[i]);
            printf("\n");
            
            printf("GPU result: \n");
            print_all_params(gpuResult, size);
            printf("\n");

            printf("Host result: \n");
            print_all_params(hostResult, size);

            exit(EXIT_FAILURE);
        }
    }
}

template <int BlockSize,
          int InputSize, int InputChannels,
          int OutputSize, int OutputChannels,
          int KernelSize,
          int PoolOutputSize, int Stride>
__global__ void convolution_gpu_shared_memory_2_maxpooling_2x2(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias,
    float* devPoolOutput)
{
    /* Assumptions: blockDim.x == 4, blockDim.y == 4 */
    /* Assumptions: KernelSize == 5, Stride == 2 */

    int i;

    int ocol = threadIdx.x + blockIdx.x * blockDim.x;
    int orow = threadIdx.y + blockIdx.y * blockDim.y;
    int och = blockIdx.z;
    int ich = threadIdx.z;

    int icol;
    int irow;
    
    int kcol;
    int krow;
    
    /* const int outputIdx = och * OutputSize * OutputSize + orow * OutputSize + ocol; */
    const int ochOffset = och * InputChannels * KernelSize * KernelSize;
    const int inputOffset = ich * InputSize * InputSize;
    const int kernelOffset = ich * KernelSize * KernelSize;
    const int tmpOffset = inputOffset + orow * InputSize + ocol;
    
    float* pWeight = devWeight + ochOffset;
    float tmp = 0.0f;
    float sum;

    const int KernelRadius = KernelSize / 2;
    const int SharedInputSize = BlockSize + KernelRadius * 2;

    __shared__ float sharedInput[InputChannels][SharedInputSize][SharedInputSize];
    __shared__ float sharedWeight[InputChannels][KernelSize][KernelSize];
    __shared__ float sharedResult[InputChannels][BlockSize][BlockSize];

    if (ocol >= OutputSize || orow >= OutputSize)
        return;

    sharedInput[ich][threadIdx.y][threadIdx.x] =
        // devInput[inputOffset + irow * InputSize + icol];
        devInput[tmpOffset];

    icol = ocol + KernelRadius * 2;

    if (icol < InputSize)
        sharedInput[ich][threadIdx.y][threadIdx.x + KernelRadius * 2] =
            // devInput[inputOffset + irow * InputSize + icol];
            devInput[tmpOffset + KernelRadius * 2];
    
    irow = orow + KernelRadius * 2;

    if (irow < InputSize)
        sharedInput[ich][threadIdx.y + KernelRadius * 2][threadIdx.x] =
            // devInput[inputOffset + irow * InputSize + icol];
            devInput[tmpOffset + InputSize * KernelRadius * 2];
    
    icol = ocol + KernelRadius * 2;
    irow = orow + KernelRadius * 2;

    if (icol < InputSize && irow < InputSize)
        sharedInput[ich][threadIdx.y + KernelRadius * 2][threadIdx.x + KernelRadius * 2] =
            // devInput[inputOffset + irow * InputSize + icol];
            devInput[tmpOffset + InputSize * KernelRadius * 2 + KernelRadius * 2];
    
    /*
     * Hack: this code works because KernelSize is 5,
     * blockDim.x is 4, and blockDim.y is also 4
     */
    sharedWeight[ich][threadIdx.y][threadIdx.x] =
        pWeight[kernelOffset + threadIdx.y * KernelSize + threadIdx.x];
    sharedWeight[ich][threadIdx.y][threadIdx.x + 1] =
        pWeight[kernelOffset + threadIdx.y * KernelSize + threadIdx.x + 1];
    sharedWeight[ich][threadIdx.y + 1][threadIdx.x] =
        pWeight[kernelOffset + (threadIdx.y + 1) * KernelSize + threadIdx.x];
    sharedWeight[ich][threadIdx.y + 1][threadIdx.x + 1] =
        pWeight[kernelOffset + (threadIdx.y + 1) * KernelSize + threadIdx.x + 1];

    __syncthreads();
    
    // sharedResult[ich][threadIdx.y][threadIdx.x] = 0.0f;
    
    #pragma unroll
    for (krow = 0; krow < KernelSize; ++krow)
        #pragma unroll
        for (kcol = 0; kcol < KernelSize; ++kcol)
            // sharedResult[ich][threadIdx.y][threadIdx.x] +=
            tmp += 
                sharedWeight[ich][krow][kcol] *
                sharedInput[ich][threadIdx.y + krow][threadIdx.x + kcol];
    
    sharedResult[ich][threadIdx.y][threadIdx.x] = tmp;

    __syncthreads();

    if (ich == 0) {
        sum = devBias[och];
        
        #pragma unroll
        for (i = 0; i < InputChannels; ++i)
            sum += sharedResult[i][threadIdx.y][threadIdx.x];

        /* devOutput[outputIdx] = sum; */
        sharedResult[0][threadIdx.y][threadIdx.x] = sum;
    }

    /* Max pooling */
    if (ich == 0 && threadIdx.x % 2 == 0 && threadIdx.y % 2 == 0) {
        float tmp[6];

        tmp[0] = sharedResult[0][threadIdx.y][threadIdx.x];
        tmp[1] = sharedResult[0][threadIdx.y][threadIdx.x + 1];
        tmp[2] = sharedResult[0][threadIdx.y + 1][threadIdx.x];
        tmp[3] = sharedResult[0][threadIdx.y + 1][threadIdx.x + 1];

        tmp[4] = fmaxf(tmp[0], tmp[1]);
        tmp[5] = fmaxf(tmp[2], tmp[3]);
        
        devPoolOutput[och * PoolOutputSize * PoolOutputSize +
                      (orow / 2) * PoolOutputSize +
                      (ocol / 2)] = fmaxf(tmp[4], tmp[5]);
    }

}

template <int BlockSize, int InputSize, int OutputSize>
__global__ void classifier_gpu_blocked_and_relu_template_2(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias)
{
    int i;
    int k;

    int weightIdxBegin = InputSize * (BlockSize * blockIdx.y);
    int outputIdx = threadIdx.y + blockDim.y * blockIdx.y;
    
    float* pWeight = devWeight + weightIdxBegin;
    float tmp = 0.0f;

    __shared__ float subInput[BlockSize][BlockSize];
    __shared__ float subWeight[BlockSize][BlockSize];
    
    for (i = 0; i < InputSize; i += BlockSize) {
        /* This implementation wastes so many threads */
        // if (threadIdx.x == 0)
            if (i + threadIdx.y < InputSize)
                subInput[0][threadIdx.y] = devInput[i + threadIdx.y];
            else
                subInput[0][threadIdx.y] = 0.0f;

        subWeight[threadIdx.y][threadIdx.x] =
            // devWeight[i + InputSize * threadIdx.y + threadIdx.x];
            pWeight[i + InputSize * threadIdx.y + threadIdx.x];

        __syncthreads();
        
        // if (threadIdx.x == 0)
        #pragma unroll
        for (k = 0; k < BlockSize; ++k)
            tmp += subWeight[threadIdx.y][k] * subInput[0][k];
    }
    
    if (threadIdx.x == 0 && outputIdx < OutputSize)
        if (tmp > 0)
            devOutput[outputIdx] = tmp;
        else
            devOutput[outputIdx] = 0.0f;
}

template <int BlockSize, int InputSize, int OutputSize>
__global__ void classifier_gpu_blocked_and_softmax_template(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias)
{
    int i;
    int k;

    int weightIdxBegin = InputSize * (BlockSize * blockIdx.y);
    int outputIdx = threadIdx.y + blockDim.y * blockIdx.y;
    
    float* pWeight = devWeight + weightIdxBegin + InputSize * threadIdx.y;
    float tmp = 0.0f;
    float sum = 0.0f;

    __shared__ float subInput[BlockSize];
    __shared__ float subOutput[OutputSize];
    
    for (i = 0; i < InputSize; i += BlockSize) {
        if (i + threadIdx.y < InputSize)
            subInput[threadIdx.y] = devInput[i + threadIdx.y];
        else
            subInput[threadIdx.y] = 0.0f;
        
        __syncthreads();
        
        #pragma unroll
        for (k = 0; k < BlockSize; ++k)
            tmp += pWeight[i + k] * subInput[k];
    }
    
    if (outputIdx < OutputSize)
        subOutput[outputIdx] = expf(tmp);

    __syncthreads();
    
    #pragma unroll
    for (k = 0; k < OutputSize; ++k)
        sum += subOutput[k];
    
    devOutput[outputIdx] = subOutput[outputIdx] / sum;
}

int main()
{
    int imageCount = 0;
    char imageFileName[64];

    float* hostImage;

    float* hostConv1Weight;
    float* hostConv1Bias;
    float* hostConv1Out;
    float* hostPool1Out;
  
    float* hostConv2Weight;
    float* hostConv2Bias;
    float* hostConv2Out;
    float* hostPool2Out;

    float* hostFc1Weight;
    float* hostFc1Bias;
    float* hostFc1Out;

    float* hostFc2Weight;
    float* hostFc2Bias;
    float* hostFc2Out;
    
    float* devImage;

    float* devConv1Weight;
    float* devConv1Bias;
    float* devConv1Out;
    float* devPool1Out;

    float* devConv2Weight;
    float* devConv2Bias;
    float* devConv2Out;
    float* devPool2Out;

    float* devFc1Weight;
    float* devFc1Bias;
    float* devFc1Out;

    float* devFc2Weight;
    float* devFc2Bias;
    float* devFc2Out;
    
    float* gpuFc2Out;

    dim3 block;
    dim3 grid;

    dim3 blockConv1(4, 4, 1);
    dim3 gridConv1(8, 8, 20);

    dim3 blockConv2(4, 4, 20);
    dim3 gridConv2(4, 4, 50);

    dim3 blockFc1(16, 16, 1);
    dim3 gridFc1(1, (500 + blockFc1.y - 1) / blockFc1.y, 1);

    dim3 blockFc2(1, 16, 1);
    dim3 gridFc2(1, 1, 1);

    hipEvent_t startEvent;
    hipEvent_t stopEvent;
    float elapsedTime;
    double gpuTimeSum = 0.0;
    double hostTimeSum = 0.0;
    
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    printf("/// LeNet ///\n");
    fflush(stdout);
    
    printf("Allocating host memory ...\n");
    fflush(stdout);

    hostImage = (float*)malloc(sizeof(float) * IMAGE_SIZE);

    hostConv1Weight = (float*)malloc(sizeof(float) * CONV1_W_SIZE);
    hostConv1Bias = (float*)malloc(sizeof(float) * CONV1_B_SIZE);
    hostConv1Out = (float*)malloc(sizeof(float) * CONV1_OUT_SIZE);
    hostPool1Out = (float*)malloc(sizeof(float) * POOL1_OUT_SIZE);
    
    hostConv2Weight = (float*)malloc(sizeof(float) * CONV2_W_SIZE);
    hostConv2Bias = (float*)malloc(sizeof(float) * CONV2_B_SIZE);
    hostConv2Out = (float*)malloc(sizeof(float) * CONV2_OUT_SIZE);
    hostPool2Out = (float*)malloc(sizeof(float) * POOL2_OUT_SIZE);

    hostFc1Weight = (float*)malloc(sizeof(float) * FC1_W_SIZE);
    hostFc1Bias = (float*)malloc(sizeof(float) * FC1_B_SIZE);
    hostFc1Out = (float*)malloc(sizeof(float) * FC1_OUT_SIZE);

    hostFc2Weight = (float*)malloc(sizeof(float) * FC2_W_SIZE);
    hostFc2Bias = (float*)malloc(sizeof(float) * FC2_B_SIZE);
    hostFc2Out = (float*)malloc(sizeof(float) * FC2_OUT_SIZE);

    gpuFc2Out = (float*)malloc(sizeof(float) * FC2_OUT_SIZE);
    
    printf("Reading parameters ...\n");
    
    /* Read Conv1 layer parameters */
    read_params("./txt/conv1_w.txt", hostConv1Weight, CONV1_W_SIZE);
    print_params("CONV1_W", hostConv1Weight, CONV1_W_SIZE);
    read_params("./txt/conv1_b.txt", hostConv1Bias, CONV1_B_SIZE);
    print_params("CONV1_B", hostConv1Bias, CONV1_B_SIZE);
    
    /* Read Conv2 layer parameters */
    read_params("./txt/conv2_w.txt", hostConv2Weight, CONV2_W_SIZE);
    print_params("CONV2_W", hostConv2Weight, CONV2_W_SIZE);
    read_params("./txt/conv2_b.txt", hostConv2Bias, CONV2_B_SIZE);
    print_params("CONV2_B", hostConv2Bias, CONV2_B_SIZE);
    
    /* Read Fc1 layer parameters */
    read_params("./txt/fc1_w.txt", hostFc1Weight, FC1_W_SIZE);
    print_params("FC1_W", hostFc1Weight, FC1_W_SIZE);
    read_params("./txt/fc1_b.txt", hostFc1Bias, FC1_B_SIZE);
    print_params("FC1_B", hostFc1Bias, FC1_B_SIZE);
    
    /* Read Fc2 layer parameters */
    read_params("./txt/fc2_w.txt", hostFc2Weight, FC2_W_SIZE);
    print_params("FC2_W", hostFc2Weight, FC2_W_SIZE);
    read_params("./txt/fc2_b.txt", hostFc2Bias, FC2_B_SIZE);
    print_params("FC2_B", hostFc2Bias, FC2_B_SIZE);
    
    printf("Allocating device memory ...\n");
    
    CUDA_SAFE_CALL(hipMalloc((void**)&devImage,
                              IMAGE_SIZE * sizeof(float)));
    
    CUDA_SAFE_CALL(hipMalloc((void**)&devConv1Weight,
                              CONV1_W_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devConv1Bias,
                              CONV1_B_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devConv1Out,
                              CONV1_OUT_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devPool1Out,
                              POOL1_OUT_SIZE * sizeof(float)));
    
    CUDA_SAFE_CALL(hipMalloc((void**)&devConv2Weight,
                              CONV2_W_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devConv2Bias,
                              CONV2_B_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devConv2Out,
                              CONV2_OUT_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devPool2Out,
                              POOL2_OUT_SIZE * sizeof(float)));
    
    CUDA_SAFE_CALL(hipMalloc((void**)&devFc1Weight,
                              FC1_W_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devFc1Bias,
                              FC1_B_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devFc1Out,
                              FC1_OUT_SIZE * sizeof(float)));
    
    CUDA_SAFE_CALL(hipMalloc((void**)&devFc2Weight,
                              FC2_W_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devFc2Bias,
                              FC2_B_SIZE * sizeof(float)));
    CUDA_SAFE_CALL(hipMalloc((void**)&devFc2Out,
                              FC2_OUT_SIZE * sizeof(float)));
    
    printf("Transferring weight and bias data from host ...\n");
    
    CUDA_SAFE_CALL(hipMemcpy(devConv1Weight, hostConv1Weight,
                              CONV1_W_SIZE * sizeof(float),
                              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(devConv1Bias, hostConv1Bias,
                              CONV1_B_SIZE * sizeof(float),
                              hipMemcpyHostToDevice));
    
    CUDA_SAFE_CALL(hipMemcpy(devConv2Weight, hostConv2Weight,
                              CONV2_W_SIZE * sizeof(float),
                              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(devConv2Bias, hostConv2Bias,
                              CONV2_B_SIZE * sizeof(float),
                              hipMemcpyHostToDevice));
    
    CUDA_SAFE_CALL(hipMemcpy(devFc1Weight, hostFc1Weight,
                              FC1_W_SIZE * sizeof(float),
                              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(devFc1Bias, hostFc1Bias,
                              FC1_B_SIZE * sizeof(float),
                              hipMemcpyHostToDevice));
    
    CUDA_SAFE_CALL(hipMemcpy(devFc2Weight, hostFc2Weight,
                              FC2_W_SIZE * sizeof(float),
                              hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(devFc2Bias, hostFc2Bias,
                              FC2_B_SIZE * sizeof(float),
                              hipMemcpyHostToDevice));

    printf("\n");

    for (imageCount = 0; imageCount < 1000; ++imageCount) {
        sprintf(imageFileName, "%simage%03d.txt", IMAGE_FILE, imageCount);

        if (imageCount % 100 == 0) {
            printf("file: %s\n", imageFileName);
            fflush(stdout);
        }

        read_params(imageFileName, hostImage, IMAGE_SIZE);
        norm_image(hostImage, IMAGE_SIZE);

        /* Feed-forward (CPU) */
        hipEventRecord(startEvent, 0);

        convolution(hostImage, 28, 1, hostConv1Out, 24, 20,
                    hostConv1Weight, hostConv1Bias, 5, 1);
        maxpooling(hostConv1Out, 24, 20, hostPool1Out, 12, 2, 2);

        convolution(hostPool1Out, 12, 20, hostConv2Out, 8, 50,
                    hostConv2Weight, hostConv2Bias, 5, 1);
        maxpooling(hostConv2Out, 8, 50, hostPool2Out, 4, 2, 2);

        classifier(hostPool2Out, 800, hostFc1Out, 500,
                   hostFc1Weight, hostFc1Bias);
        relu(hostFc1Out, 1, 500);

        classifier(hostFc1Out, 500, hostFc2Out, 10,
                   hostFc2Weight, hostFc2Bias);
        softmax(hostFc2Out, 10);

        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
        hostTimeSum += (double)elapsedTime;

        /* Feed-Forward (GPU) */
        CUDA_SAFE_CALL(hipMemcpy(devImage, hostImage,
                                  IMAGE_SIZE * sizeof(float),
                                  hipMemcpyHostToDevice));
        hipEventRecord(startEvent, 0);

        convolution_gpu_shared_memory_2_maxpooling_2x2
            <4, 28, 1, 24, 20, 5, 12, 2><<<gridConv1, blockConv1>>>(
                devImage, NULL, devConv1Weight, devConv1Bias, devPool1Out);

        convolution_gpu_shared_memory_2_maxpooling_2x2
            <4, 12, 20, 8, 50, 5, 4, 2><<<gridConv2, blockConv2>>>(
                devPool1Out, NULL, devConv2Weight, devConv2Bias, devPool2Out);

        classifier_gpu_blocked_and_relu_template_2
            <16, 800, 500><<<gridFc1, blockFc1>>>(
                devPool2Out, devFc1Out, devFc1Weight, devFc1Bias);

        classifier_gpu_blocked_and_softmax_template
            <16, 500, 10><<<gridFc2, blockFc2>>>(
                devFc1Out, devFc2Out, devFc2Weight, devFc2Bias);

        CUDA_SAFE_CALL(hipMemcpy(gpuFc2Out, devFc2Out,
                                  FC2_OUT_SIZE * sizeof(float),
                                  hipMemcpyDeviceToHost));

        hipEventRecord(stopEvent, 0);
        hipEventSynchronize(stopEvent);
        hipEventElapsedTime(&elapsedTime, startEvent, stopEvent);
        gpuTimeSum += elapsedTime;

        check_result(hostFc2Out, gpuFc2Out, 10);
    }

    printf("GPU implementation is %f times faster than CPU\n",
           hostTimeSum / gpuTimeSum);
    printf("Average processing time: CPU: %f ms, GPU: %f ms\n",
           hostTimeSum / 1000.0f, gpuTimeSum / 1000.0f);

    /* Free device memory */
    CUDA_SAFE_CALL(hipFree(devImage));

    CUDA_SAFE_CALL(hipFree(devConv1Weight));
    CUDA_SAFE_CALL(hipFree(devConv1Bias));
    CUDA_SAFE_CALL(hipFree(devConv1Out));
    CUDA_SAFE_CALL(hipFree(devPool1Out));
    
    CUDA_SAFE_CALL(hipFree(devConv2Weight));
    CUDA_SAFE_CALL(hipFree(devConv2Bias));
    CUDA_SAFE_CALL(hipFree(devConv2Out));
    CUDA_SAFE_CALL(hipFree(devPool2Out));

    CUDA_SAFE_CALL(hipFree(devFc1Weight));
    CUDA_SAFE_CALL(hipFree(devFc1Bias));
    CUDA_SAFE_CALL(hipFree(devFc1Out));

    CUDA_SAFE_CALL(hipFree(devFc2Weight));
    CUDA_SAFE_CALL(hipFree(devFc2Bias));
    CUDA_SAFE_CALL(hipFree(devFc2Out));

    /* Free host memory */
    free(hostImage);

    free(hostConv1Weight);
    free(hostConv1Bias);
    free(hostConv1Out);
    free(hostPool1Out);
    
    free(hostConv2Weight);
    free(hostConv2Bias);
    free(hostConv2Out);
    free(hostPool2Out);

    free(hostFc1Weight);
    free(hostFc1Bias);
    free(hostFc1Out);
    
    free(hostFc2Weight);
    free(hostFc2Bias);
    free(hostFc2Out);

    free(gpuFc2Out);

    /* Reset device */
    CUDA_SAFE_CALL(hipDeviceReset());

    return EXIT_SUCCESS;
}

