
#include <hip/hip_runtime.h>

/* lenet_old.cu */

template <int InputSize, int InputChannels,
          int OutputSize, int OutputChannels,
          int KernelSize>
__global__ void convolution_gpu_shared_memory(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias)
{
    int ocol = threadIdx.x + blockIdx.x * blockDim.x;
    int orow = threadIdx.y + blockIdx.y * blockDim.y;
    int och = blockIdx.z;

    int icol;
    int irow;
    
    int kcol;
    int krow;
    int kch;
    
    int outputIdx = och * OutputSize * OutputSize + orow * OutputSize + ocol;
    int ochOffset = och * InputChannels * KernelSize * KernelSize;

    float* pWeight = devWeight + ochOffset;
    float sum;

    __shared__ float sharedInput[InputChannels][InputSize][InputSize];

    if (ocol >= OutputSize || orow >= OutputSize || och >= OutputChannels)
        return;

    icol = ocol;
    irow = orow;
    
    for (kch = 0; kch < InputChannels; ++kch)
        sharedInput[kch][irow][icol] =
            devInput[kch * InputSize * InputSize + irow * InputSize + icol];

    icol = ocol + KernelSize;
    irow = orow;

    if (icol < InputSize)
        for (kch = 0; kch < InputChannels; ++kch)
            sharedInput[kch][irow][icol] =
                devInput[kch * InputSize * InputSize + irow * InputSize + icol];
    
    icol = ocol;
    irow = orow + KernelSize;

    if (irow < InputSize)
        for (kch = 0; kch < InputChannels; ++kch)
            sharedInput[kch][irow][icol] =
                devInput[kch * InputSize * InputSize + irow * InputSize + icol];
    
    icol = ocol + KernelSize;
    irow = orow + KernelSize;

    if (icol < InputSize && irow < InputSize)
        for (kch = 0; kch < InputChannels; ++kch)
            sharedInput[kch][irow][icol] =
                devInput[kch * InputSize * InputSize + irow * InputSize + icol];

    __syncthreads();
    
    sum = devBias[och];

    for (kch = 0; kch < InputChannels; ++kch)
        for (krow = 0; krow < KernelSize; ++krow)
            for (kcol = 0; kcol < KernelSize; ++kcol)
                sum += pWeight[kch * KernelSize * KernelSize + krow * KernelSize + kcol] *
                       sharedInput[kch][orow + krow][ocol + kcol];

    devOutput[outputIdx] = sum;
}

template <int BlockSize,
          int InputSize, int InputChannels,
          int OutputSize, int OutputChannels,
          int KernelSize>
__global__ void convolution_gpu_shared_memory_2(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias)
{
    int i;

    int ocol = threadIdx.x + blockIdx.x * blockDim.x;
    int orow = threadIdx.y + blockIdx.y * blockDim.y;
    int och = blockIdx.z;
    int ich = threadIdx.z;

    int icol;
    int irow;
    
    int kcol;
    int krow;
    
    const int outputIdx = och * OutputSize * OutputSize + orow * OutputSize + ocol;
    const int ochOffset = och * InputChannels * KernelSize * KernelSize;
    const int inputOffset = ich * InputSize * InputSize;
    const int kernelOffset = ich * KernelSize * KernelSize;
    
    float* pWeight = devWeight + ochOffset;
    float sum;

    const int KernelRadius = KernelSize / 2;
    const int SharedInputSize = BlockSize + KernelRadius * 2;

    __shared__ float sharedInput[InputChannels][SharedInputSize][SharedInputSize];
    __shared__ float sharedWeight[InputChannels][KernelSize][KernelSize];
    __shared__ float sharedResult[InputChannels][BlockSize][BlockSize];

    if (ocol >= OutputSize || orow >= OutputSize)
        return;

    icol = ocol;
    irow = orow;

    sharedInput[ich][threadIdx.y][threadIdx.x] =
        devInput[inputOffset + irow * InputSize + icol];

    icol = ocol + KernelRadius * 2;
    irow = orow;

    if (icol < InputSize)
        sharedInput[ich][threadIdx.y][threadIdx.x + KernelRadius * 2] =
            devInput[inputOffset + irow * InputSize + icol];
    
    icol = ocol;
    irow = orow + KernelRadius * 2;

    if (irow < InputSize)
        sharedInput[ich][threadIdx.y + KernelRadius * 2][threadIdx.x] =
            devInput[inputOffset + irow * InputSize + icol];
    
    icol = ocol + KernelRadius * 2;
    irow = orow + KernelRadius * 2;

    if (icol < InputSize && irow < InputSize)
        sharedInput[ich][threadIdx.y + KernelRadius * 2][threadIdx.x + KernelRadius * 2] =
            devInput[inputOffset + irow * InputSize + icol];
    
    /*
     * Hack: this code works because KernelSize is 5,
     * blockDim.x is 4, and blockDim.y is also 4
     */
    sharedWeight[ich][threadIdx.y][threadIdx.x] =
        pWeight[kernelOffset + threadIdx.y * KernelSize + threadIdx.x];
    sharedWeight[ich][threadIdx.y][threadIdx.x + 1] =
        pWeight[kernelOffset + threadIdx.y * KernelSize + threadIdx.x + 1];
    sharedWeight[ich][threadIdx.y + 1][threadIdx.x] =
        pWeight[kernelOffset + (threadIdx.y + 1) * KernelSize + threadIdx.x];
    sharedWeight[ich][threadIdx.y + 1][threadIdx.x + 1] =
        pWeight[kernelOffset + (threadIdx.y + 1) * KernelSize + threadIdx.x + 1];

    __syncthreads();
    
    sharedResult[ich][threadIdx.y][threadIdx.x] = 0.0f;
    
    for (krow = 0; krow < KernelSize; ++krow)
        for (kcol = 0; kcol < KernelSize; ++kcol)
            sharedResult[ich][threadIdx.y][threadIdx.x] +=
                sharedWeight[ich][krow][kcol] *
                sharedInput[ich][threadIdx.y + krow][threadIdx.x + kcol];
    
    __syncthreads();

    if (ich == 0) {
        sum = devBias[och];

        for (i = 0; i < InputChannels; ++i)
            sum += sharedResult[i][threadIdx.y][threadIdx.x];

        devOutput[outputIdx] = sum;
    }
}

template <int InputSize, int InputChannels,
          int OutputSize, int Stride>
__global__ void maxpooling_gpu_kernel_2x2_template(
    float* devInput, float* devOutput)
{
    int ocol = threadIdx.x;
    int orow = threadIdx.y;
    int och = blockIdx.z;

    float tmp0;
    float tmp1;
    float tmp2;
    float tmp3;
    float tmp4;
    float tmp5;

    int outputIdx = och * OutputSize * OutputSize + orow * OutputSize + ocol;
    int inputOffset = och * InputSize * InputSize +
                      (orow * Stride) * InputSize +
                      (ocol * Stride);
    
    if (ocol >= OutputSize || orow >= OutputSize || och >= InputChannels)
        return;

    tmp0 = devInput[inputOffset];
    tmp1 = devInput[inputOffset + 1];
    tmp2 = devInput[inputOffset + InputSize];
    tmp3 = devInput[inputOffset + InputSize + 1];

    tmp4 = fmaxf(tmp0, tmp1);
    tmp5 = fmaxf(tmp2, tmp3);

    devOutput[outputIdx] = fmaxf(tmp4, tmp5);
}

template <int BlockSize, int InputSize, int OutputSize>
__global__ void classifier_gpu_blocked_and_relu_template(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias)
{
    int i;
    int j;
    int k;

    int weightIdxBegin = InputSize * (BlockSize * blockIdx.y);
    int weightIdxEnd = weightIdxBegin + InputSize;
    int outputIdx = threadIdx.y + blockDim.y * blockIdx.y;

    float tmp = 0.0f;

    __shared__ float subInput[BlockSize];
    
    for (i = weightIdxBegin, j = 0; i < weightIdxEnd;
         i += BlockSize, j += BlockSize) {
        if (j + threadIdx.y < InputSize)
            subInput[threadIdx.y] = devInput[j + threadIdx.y];
        else
            subInput[threadIdx.y] = 0.0f;

        __syncthreads();
        
        #pragma unroll
        for (k = 0; k < BlockSize; ++k)
            tmp += devWeight[i + InputSize * threadIdx.y + k] * subInput[k];

        __syncthreads();
    }

    if (outputIdx < OutputSize)
        if (tmp > 0.0f)
            devOutput[outputIdx] = tmp;
        else
            devOutput[outputIdx] = 0.0f;
}

template <int BlockSize, int InputSize, int OutputSize>
__global__ void classifier_gpu_blocked_and_relu_template_3(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias)
{
    int k;
    int outputIdx = threadIdx.y + blockDim.y * blockIdx.y;

    float* pInput = devInput + BlockSize * threadIdx.x;
    float* pWeight = devWeight + InputSize * outputIdx + BlockSize * threadIdx.x;
    float tmp = 0.0f;
    
    __shared__ float subOutput[BlockSize][InputSize / BlockSize];

    #pragma unroll
    for (k = 0; k < BlockSize; ++k)
        tmp += pWeight[k] * pInput[k];

    subOutput[threadIdx.y][threadIdx.x] = tmp;
    __syncthreads();

    if (threadIdx.x == 0) {
        #pragma unroll
        for (k = 1; k < InputSize / BlockSize; ++k)
            subOutput[threadIdx.y][0] += subOutput[threadIdx.y][k];

        subOutput[threadIdx.y][0] += devBias[outputIdx];

        devOutput[outputIdx] = subOutput[threadIdx.y][0] * (subOutput[threadIdx.y][0] > 0);
    }
}

template <int BlockSize, int InputSize, int OutputSize>
__global__ void classifier_gpu_blocked_and_softmax_template(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias)
{
    int i;
    int k;

    // int weightIdxBegin = InputSize * (BlockSize * blockIdx.y);
    // int outputIdx = threadIdx.y + blockDim.y * blockIdx.y;
    int weightIdxBegin = 0;
    int outputIdx = threadIdx.y;
    
    // float* pWeight = devWeight + weightIdxBegin + InputSize * threadIdx.y;
    float* pWeight = devWeight + InputSize * threadIdx.y;
    float tmp = 0.0f;
    float sum = 0.0f;

    __shared__ float subInput[BlockSize];
    __shared__ float subOutput[OutputSize];
    
    for (i = 0; i < InputSize; i += BlockSize) {
        if (i + threadIdx.y < InputSize)
            subInput[threadIdx.y] = devInput[i + threadIdx.y];
        else
            subInput[threadIdx.y] = 0.0f;
        
        __syncthreads();
        
        #pragma unroll
        for (k = 0; k < BlockSize; ++k)
            tmp += pWeight[i + k] * subInput[k];
    }
    
    if (outputIdx < OutputSize)
        subOutput[outputIdx] = expf(tmp);

    __syncthreads();
    
    #pragma unroll
    for (k = 0; k < OutputSize; ++k)
        sum += subOutput[k];
    
    if (outputIdx < OutputSize)
        devOutput[outputIdx] = subOutput[outputIdx] / sum;
}

template <int BlockSize,
          int InputSize, int InputChannels,
          int OutputSize, int OutputChannels,
          int PoolOutputSize>
__global__ void convolution_gpu_shared_memory_2_maxpooling_2x2_old(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias,
    float* devPoolOutput)
{
    /* Assumptions: blockDim.x == 4, blockDim.y == 4 */
    
    const int KernelSize = 5;

    const int ocol = threadIdx.x + blockIdx.x * blockDim.x;
    const int orow = threadIdx.y + blockIdx.y * blockDim.y;
    const int och = blockIdx.z;
    const int ich = threadIdx.z;
    
    /* const int outputIdx = och * OutputSize * OutputSize + orow * OutputSize + ocol; */
    const int ochOffset = och * InputChannels * KernelSize * KernelSize;
    const int inputOffset = ich * InputSize * InputSize;
    const int kernelOffset = ich * KernelSize * KernelSize;
    const int tmpOffset = inputOffset + orow * InputSize + ocol;
    
    const int KernelRadius = KernelSize / 2;
    const int SharedInputSize = BlockSize + KernelRadius * 2;

    int i;
    int icol;
    int irow;
    int kcol;
    int krow;

    float* pWeight = devWeight + ochOffset;
    float tmp = 0.0f;
    float sum = 0.0f;
    
    __shared__ float sharedInput[InputChannels][SharedInputSize][SharedInputSize];
    __shared__ float sharedWeight[InputChannels][KernelSize][KernelSize];
    __shared__ float sharedResult[InputChannels][BlockSize][BlockSize];

    if (ocol >= OutputSize || orow >= OutputSize)
        return;
    
    /*
     * Bring input data to shared memory
     */
    sharedInput[ich][threadIdx.y][threadIdx.x] =
        devInput[tmpOffset];

    icol = ocol + KernelRadius * 2;

    if (icol < InputSize)
        sharedInput[ich][threadIdx.y][threadIdx.x + KernelRadius * 2] =
            devInput[tmpOffset + KernelRadius * 2];
    
    irow = orow + KernelRadius * 2;

    if (irow < InputSize)
        sharedInput[ich][threadIdx.y + KernelRadius * 2][threadIdx.x] =
            devInput[tmpOffset + InputSize * KernelRadius * 2];

    if (icol < InputSize && irow < InputSize)
        sharedInput[ich][threadIdx.y + KernelRadius * 2][threadIdx.x + KernelRadius * 2] =
            devInput[tmpOffset + InputSize * KernelRadius * 2 + KernelRadius * 2];
    
    /*
     * Bring weight data to shared memory
     */

    /*
     * Hack: this code works because KernelSize == 5,
     * blockDim.x == blockDim.y == 4
     */
    pWeight += kernelOffset + threadIdx.y * KernelSize + threadIdx.x;
    sharedWeight[ich][threadIdx.y][threadIdx.x] = *pWeight;
    sharedWeight[ich][threadIdx.y][threadIdx.x + 1] = *(pWeight + 1);

    pWeight += KernelSize;
    sharedWeight[ich][threadIdx.y + 1][threadIdx.x] = *pWeight;
    sharedWeight[ich][threadIdx.y + 1][threadIdx.x + 1] = *(pWeight + 1);

    __syncthreads();
    
    #pragma unroll
    for (krow = 0; krow < KernelSize; ++krow)
        #pragma unroll
        for (kcol = 0; kcol < KernelSize; ++kcol)
            tmp += sharedWeight[ich][krow][kcol] *
                   sharedInput[ich][threadIdx.y + krow][threadIdx.x + kcol];
    
    sharedResult[ich][threadIdx.y][threadIdx.x] = tmp;
    __syncthreads();

    if (ich == 0) {
        sum = devBias[och];
        
        #pragma unroll
        for (i = 0; i < InputChannels; ++i)
            sum += sharedResult[i][threadIdx.y][threadIdx.x];

        sharedResult[0][threadIdx.y][threadIdx.x] = sum;
        __syncthreads();

        /* Max pooling */
        if (threadIdx.x % 2 == 0 && threadIdx.y % 2 == 0) {
            float tmp0;
            float tmp1;

            tmp0 = fmaxf(sharedResult[0][threadIdx.y][threadIdx.x],
                         sharedResult[0][threadIdx.y][threadIdx.x + 1]);
            tmp1 = fmaxf(sharedResult[0][threadIdx.y + 1][threadIdx.x],
                         sharedResult[0][threadIdx.y + 1][threadIdx.x + 1]);
            
            devPoolOutput[och * PoolOutputSize * PoolOutputSize +
                          (orow / 2) * PoolOutputSize + (ocol / 2)]
                          = fmaxf(tmp0, tmp1);
        }

        /* if (threadIdx.x < BlockSize / 2 && threadIdx.y < BlockSize / 2) {
            float tmp0;
            float tmp1;

            tmp0 = fmaxf(sharedResult[0][threadIdx.y * 2][threadIdx.x * 2],
                         sharedResult[0][threadIdx.y * 2][threadIdx.x * 2 + 1]);
            tmp1 = fmaxf(sharedResult[0][threadIdx.y * 2 + 1][threadIdx.x * 2],
                         sharedResult[0][threadIdx.y * 2 + 1][threadIdx.x * 2 + 1]);

            devPoolOutput[och * PoolOutputSize * PoolOutputSize +
                          ((blockDim.y * blockIdx.y) / 2 + threadIdx.y) * PoolOutputSize +
                          ((blockDim.x * blockIdx.x) / 2 + threadIdx.x)]
                          = fmaxf(tmp0, tmp1);
        } */
        
        /* if (threadIdx.x % 2 == 0) {
            sharedResult[0][threadIdx.y][threadIdx.x] = fmaxf(
                sharedResult[0][threadIdx.y][threadIdx.x],
                sharedResult[0][threadIdx.y][threadIdx.x + 1]);
        }

        __syncthreads();

        if (threadIdx.y % 2 == 0) {
            sharedResult[0][threadIdx.y][threadIdx.x] = fmaxf(
                sharedResult[0][threadIdx.y][threadIdx.x],
                sharedResult[0][threadIdx.y + 1][threadIdx.x]);
        }

        __syncthreads();
        
        if (threadIdx.x % 2 == 0 && threadIdx.y % 2 == 0) {
            devPoolOutput[och * PoolOutputSize * PoolOutputSize +
                          (orow / 2) * PoolOutputSize + (ocol / 2)] =
                sharedResult[0][threadIdx.y][threadIdx.x];
        } */
    }
}

template <int BlockSize, int InputSize, int OutputSize>
__global__ void classifier_gpu_blocked_and_relu_template_2(
    float* devInput, float* devOutput,
    float* devWeight, float* devBias)
{
    int i;
    int k;

    int weightIdxBegin = InputSize * (BlockSize * blockIdx.y);
    int outputIdx = threadIdx.y + blockDim.y * blockIdx.y;
    
    float* pWeight = devWeight + weightIdxBegin + InputSize * threadIdx.y + threadIdx.x;
    float tmp = 0.0f;

    __shared__ float subInput[BlockSize];
    __shared__ float subWeight[BlockSize][BlockSize];
    __shared__ float subOutput[BlockSize][BlockSize];

    subOutput[threadIdx.y][threadIdx.x] = 0.0f;
    __syncthreads();
    
    for (i = 0; i < InputSize; i += BlockSize) {
        /* This implementation wastes so many threads */
        subInput[threadIdx.y] = devInput[i + threadIdx.y];
        subWeight[threadIdx.y][threadIdx.x] = pWeight[i];
        __syncthreads();

        subOutput[threadIdx.y][threadIdx.x] +=
            subWeight[threadIdx.y][threadIdx.x] * subInput[threadIdx.x];
    }

    __syncthreads();

    // if (threadIdx.x == 0 && outputIdx < OutputSize) {
    if (threadIdx.x == 0) {
        #pragma unroll 4
        for (k = 0; k < BlockSize; ++k)
            tmp += subOutput[threadIdx.y][k];
        
        tmp += devBias[outputIdx];
        devOutput[outputIdx] = fmaxf(tmp, 0.0f);
    }
}

